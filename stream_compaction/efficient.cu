#include "hip/hip_runtime.h"
#include "common.h"
#include "efficient.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <memory>
#include <optional>

namespace StreamCompaction {
namespace Efficient {

using StreamCompaction::Common::PerformanceTimer;

/// Number of threads per block.
constexpr int blockSize = 256;

/// Enable `checkCUDAError()` calls within the performance measuring fence.
constexpr bool checkErrorsDuringTimer = true;

PerformanceTimer& timer() {
  static PerformanceTimer timer;
  return timer;
}

__global__ void kernReduceForLayer(int n, int* data, int layer, int stride) {
  int k = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (k >= n) return;

  int offset = k * stride;
  int previousStride = 1 << layer;

  int rightChild = offset + stride - 1;
  int leftChild = offset + previousStride - 1;

  data[rightChild] += data[leftChild];
}

__global__ void kernTraverseDownLayer(int n, int* data, int layer, int stride) {
  int k = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (k >= n) return;

  int offset = k * stride;
  int previousStride = 1 << layer;

  int rightChild = offset + stride - 1;
  int leftChild = offset + previousStride - 1;

  int leftValue = data[leftChild];
  data[leftChild] = data[rightChild];
  data[rightChild] += leftValue;
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int* odata, const int* idata) {
  int actualN = n;
  size_t numBytes = n * sizeof(int);

  std::unique_ptr<int[]> actualInputData = std::make_unique<int[]>(n);
  std::memcpy(actualInputData.get(), idata, numBytes);

  // Input array size is not a power of two; we have to pad the left with zeroes
  std::optional<int> paddingOpt;
  if (int numLeaves = 1 << ilog2ceil(n); n < numLeaves) {
    int offset = numLeaves - n;

    // Pad to the next power of two
    std::unique_ptr<int[]> paddedInputData = std::make_unique<int[]>(numLeaves);
    std::memcpy(paddedInputData.get() + offset, idata, numBytes);

    paddingOpt = offset;
    actualN = numLeaves;
    numBytes = numLeaves * sizeof(int);
    actualInputData.swap(paddedInputData);
  }

  int* dev_data = nullptr;
  hipMalloc(reinterpret_cast<void**>(&dev_data), numBytes);
  checkCUDAError("hipMalloc: dev_data failed!");
  hipMemcpy(dev_data, actualInputData.get(), numBytes, hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy: actualInputData -> dev_data failed!");

  timer().startGpuTimer();

  // Perform up-sweep via parallel reduction
  for (int layer = 0; layer < ilog2(actualN); ++layer) {
    int stride = 1 << (layer + 1);
    int numDispatches = actualN / stride;
    int numBlocks = (numDispatches + blockSize - 1) / blockSize;

    kernReduceForLayer<<<numBlocks, blockSize>>>(numDispatches, dev_data, layer, stride);
  }

  // Zero out the root
  int zero = 0;
  hipMemcpy(dev_data + (actualN - 1), &zero, sizeof(int), hipMemcpyHostToDevice);
  if constexpr (checkErrorsDuringTimer) checkCUDAError("hipMemcpy: 0 -> dev_data failed!");

  for (int layer = ilog2(actualN) - 1; layer >= 0; --layer) {
    int stride = 1 << (layer + 1);
    int numDispatches = actualN / stride;
    int numBlocks = (numDispatches + blockSize - 1) / blockSize;

    kernTraverseDownLayer<<<numBlocks, blockSize>>>(numDispatches, dev_data, layer, stride);
  }

  timer().endGpuTimer();

  if (paddingOpt) {
    // If previously padded, remove extra zeroes
    hipMemcpy(actualInputData.get(), dev_data, numBytes, hipMemcpyDeviceToHost);
    checkCUDAError("hipMemcpy: dev_data -> actualInputData failed!");
    std::memcpy(odata, actualInputData.get() + paddingOpt.value(), n * sizeof(int));
  } else {
    hipMemcpy(odata, dev_data, numBytes, hipMemcpyDeviceToHost);
    checkCUDAError("hipMemcpy: dev_data -> odata failed!");
  }

  hipFree(dev_data);
  checkCUDAError("hipFree: dev_data failed!");
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int* odata, const int* idata) {
  timer().startGpuTimer();
  // TODO
  timer().endGpuTimer();
  return -1;
}

}  // namespace Efficient
}  // namespace StreamCompaction
