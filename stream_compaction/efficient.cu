#include "hip/hip_runtime.h"
#include "common.h"
#include "efficient.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <memory>
#include <optional>

namespace StreamCompaction {
namespace Efficient {

namespace {

bool enableScanMeasure = true;

}

using StreamCompaction::Common::PerformanceTimer;

/// Number of threads per block.
constexpr int blockSize = 256;

/// Enable `checkCUDAError()` calls within the performance measuring fence.
constexpr bool checkErrorsDuringTimer = true;

PerformanceTimer& timer() {
  static PerformanceTimer timer;
  return timer;
}

__global__ void kernReduceForLayer(int n, int* data, int layer, int stride) {
  int k = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (k >= n) return;

  int offset = k * stride;
  int previousStride = 1 << layer;

  int rightChild = offset + stride - 1;
  int leftChild = offset + previousStride - 1;

  data[rightChild] += data[leftChild];
}

__global__ void kernTraverseDownLayer(int n, int* data, int layer, int stride) {
  int k = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (k >= n) return;

  int offset = k * stride;
  int previousStride = 1 << layer;

  int rightChild = offset + stride - 1;
  int leftChild = offset + previousStride - 1;

  int leftValue = data[leftChild];
  data[leftChild] = data[rightChild];
  data[rightChild] += leftValue;
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int* odata, const int* idata) {
  int actualN = n;
  size_t numBytes = n * sizeof(int);

  std::unique_ptr<int[]> actualInputData = std::make_unique<int[]>(n);
  std::memcpy(actualInputData.get(), idata, numBytes);

  // Input array size is not a power of two; we have to pad the left with zeroes
  std::optional<int> paddingOpt;
  if (int numLeaves = 1 << ilog2ceil(n); n < numLeaves) {
    int offset = numLeaves - n;

    // Pad to the next power of two
    std::unique_ptr<int[]> paddedInputData = std::make_unique<int[]>(numLeaves);
    std::memcpy(paddedInputData.get() + offset, idata, numBytes);

    paddingOpt = offset;
    actualN = numLeaves;
    numBytes = numLeaves * sizeof(int);
    actualInputData.swap(paddedInputData);
  }

  int* dev_data = nullptr;
  hipMalloc(reinterpret_cast<void**>(&dev_data), numBytes);
  checkCUDAError("hipMalloc: dev_data failed!");
  hipMemcpy(dev_data, actualInputData.get(), numBytes, hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy: actualInputData -> dev_data failed!");

  if (enableScanMeasure) timer().startGpuTimer();

  // Perform up-sweep via parallel reduction
  for (int layer = 0; layer < ilog2(actualN); ++layer) {
    int stride = 1 << (layer + 1);
    int numDispatches = actualN / stride;
    int numBlocks = (numDispatches + blockSize - 1) / blockSize;

    kernReduceForLayer<<<numBlocks, blockSize>>>(numDispatches, dev_data, layer, stride);
  }

  // Zero out the root
  int zero = 0;
  hipMemcpy(dev_data + (actualN - 1), &zero, sizeof(int), hipMemcpyHostToDevice);
  if constexpr (checkErrorsDuringTimer) checkCUDAError("hipMemcpy: 0 -> dev_data failed!");

  for (int layer = ilog2(actualN) - 1; layer >= 0; --layer) {
    int stride = 1 << (layer + 1);
    int numDispatches = actualN / stride;
    int numBlocks = (numDispatches + blockSize - 1) / blockSize;

    kernTraverseDownLayer<<<numBlocks, blockSize>>>(numDispatches, dev_data, layer, stride);
  }

  if (enableScanMeasure) timer().endGpuTimer();

  if (paddingOpt) {
    // If previously padded, remove extra zeroes
    hipMemcpy(actualInputData.get(), dev_data, numBytes, hipMemcpyDeviceToHost);
    checkCUDAError("hipMemcpy: dev_data -> actualInputData failed!");
    std::memcpy(odata, actualInputData.get() + paddingOpt.value(), n * sizeof(int));
  } else {
    hipMemcpy(odata, dev_data, numBytes, hipMemcpyDeviceToHost);
    checkCUDAError("hipMemcpy: dev_data -> odata failed!");
  }

  hipFree(dev_data);
  checkCUDAError("hipFree: dev_data failed!");
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int* odata, const int* idata) {
  int* dev_bools = nullptr;
  int* dev_indices = nullptr;
  int* dev_odata = nullptr;
  int* dev_idata = nullptr;

  int numBlocks = (n + blockSize - 1) / blockSize;
  size_t numBytes = n * sizeof(int);

  hipMalloc(reinterpret_cast<void**>(&dev_bools), numBytes);
  checkCUDAError("hipMalloc: dev_bools failed");
  hipMalloc(reinterpret_cast<void**>(&dev_indices), numBytes);
  checkCUDAError("hipMalloc: dev_indices failed");
  hipMalloc(reinterpret_cast<void**>(&dev_odata), numBytes);
  checkCUDAError("hipMalloc: dev_odata failed");
  hipMalloc(reinterpret_cast<void**>(&dev_idata), numBytes);
  checkCUDAError("hipMalloc: dev_idata failed");

  std::unique_ptr<int[]> indices = std::make_unique<int[]>(n);
  std::unique_ptr<int[]> bools = std::make_unique<int[]>(n);

  hipMemcpy(dev_idata, idata, numBytes, hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy: idata -> dev_idata failed");

  timer().startGpuTimer();

  Common::kernMapToBoolean<<<numBlocks, blockSize>>>(n, dev_bools, dev_idata);
  hipMemcpy(bools.get(), dev_bools, numBytes, hipMemcpyDeviceToHost);
  if constexpr (checkErrorsDuringTimer) checkCUDAError("hipMemcpy: dev_bools -> bools failed");

  enableScanMeasure = false;
  scan(n, indices.get(), bools.get());
  enableScanMeasure = true;

  hipMemcpy(dev_indices, indices.get(), numBytes, hipMemcpyHostToDevice);
  if constexpr (checkErrorsDuringTimer) checkCUDAError("hipMemcpy: indices -> dev_indices failed");
  Common::kernScatter<<<numBlocks, blockSize>>>(n, dev_odata, dev_idata, dev_bools, dev_indices);

  timer().endGpuTimer();

  hipMemcpy(odata, dev_indices, numBytes, hipMemcpyDeviceToHost);
  checkCUDAError("hipMemcpy: dev_indices -> odata failed");
  int numRemaining = odata[n - 1];

  hipMemcpy(odata, dev_odata, numBytes, hipMemcpyDeviceToHost);
  checkCUDAError("hipMemcpy: dev_odata -> odata failed");

  hipFree(dev_bools);
  checkCUDAError("hipFree: dev_bools failed");
  hipFree(dev_indices);
  checkCUDAError("hipFree: dev_indices failed");
  hipFree(dev_odata);
  checkCUDAError("hipFree: dev_odata failed");
  hipFree(dev_idata);
  checkCUDAError("hipFree: dev_idata failed");

  // Since we're doing an exclusive scan, we need to manually check if the last element is valid
  if (bools[n - 1]) {
    return numRemaining + 1;
  } else {
    return numRemaining;
  }
}

}  // namespace Efficient
}  // namespace StreamCompaction
