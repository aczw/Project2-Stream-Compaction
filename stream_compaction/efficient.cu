#include "common.h"
#include "efficient.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <memory>

namespace StreamCompaction {
namespace Efficient {

using StreamCompaction::Common::PerformanceTimer;

PerformanceTimer& timer() {
  static PerformanceTimer timer;
  return timer;
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int* odata, const int* idata) {
  int actualN = n;
  const int* actualInputData = idata;
  size_t numBytes = n * sizeof(int);

  // Input array size is not a power of two; we have to pad the left with zeroes
  if (int numLeaves = 1 << ilog2ceil(n); n < numLeaves) {
    int offset = numLeaves - n;

    // Pad to the next power of two
    std::unique_ptr<int[]> paddedInputData = std::make_unique<int[]>(numLeaves);
    std::memcpy(paddedInputData.get() + offset, idata, numBytes);

    actualN = numLeaves;
    actualInputData = paddedInputData.release();
    numBytes = numLeaves * sizeof(int);
  }

  int* dev_data = nullptr;
  hipMalloc(reinterpret_cast<void**>(&dev_data), numBytes);
  checkCUDAError("hipMalloc: dev_data failed!");
  hipMemcpy(dev_data, actualInputData, numBytes, hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy: actualInputData -> dev_data failed!");

  timer().startGpuTimer();
  // TODO
  timer().endGpuTimer();

  hipFree(dev_data);
  checkCUDAError("hipFree: dev_data failed!");
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int* odata, const int* idata) {
  timer().startGpuTimer();
  // TODO
  timer().endGpuTimer();
  return -1;
}

}  // namespace Efficient
}  // namespace StreamCompaction
