#include "hip/hip_runtime.h"
#include "common.h"
#include "naive.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace StreamCompaction {
namespace Naive {

using StreamCompaction::Common::PerformanceTimer;

/// Number of threads per block.
constexpr int blockSize = 256;

/// Whether to return an inclusive or exclusive scan.
constexpr bool useExclusiveScan = true;

PerformanceTimer& timer() {
  static PerformanceTimer timer;
  return timer;
}

__global__ void kernSumPairsForIteration(int n, const int* in, int* out, int stride) {
  int k = (blockIdx.x * blockDim.x) + threadIdx.x;

  // As the number of dispatches decrease with every iteration, we have to add the stride to
  // get the last index of the array
  if (k >= n + stride) return;

  if (k >= stride) {
    out[k] = in[k - stride] + in[k];
  }
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int* odata, const int* idata) {
  int* dev_dataA = nullptr;
  int* dev_dataB = nullptr;

  size_t numBytes = n * sizeof(int);
  hipMalloc(reinterpret_cast<void**>(&dev_dataA), numBytes);
  checkCUDAError("hipMalloc: dev_dataA failed!");
  hipMalloc(reinterpret_cast<void**>(&dev_dataB), numBytes);
  checkCUDAError("hipMalloc: dev_dataB failed!");

  hipMemcpy(dev_dataA, idata, numBytes, hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy: idata -> dev_dataA failed!");
  hipMemcpy(dev_dataB, dev_dataA, numBytes, hipMemcpyDeviceToDevice);
  checkCUDAError("hipMemcpy: dev_dataA -> dev_dataB failed!");

  timer().startGpuTimer();

  for (int iteration = 1; iteration <= ilog2ceil(n); ++iteration) {
    int stride = 1 << (iteration - 1);
    int numDispatches = n - stride;
    int numBlocks = (numDispatches + blockSize - 1) / blockSize;

    kernSumPairsForIteration<<<numBlocks, blockSize>>>(numDispatches, dev_dataA, dev_dataB, stride);

    // Write new results back into A to be read from
    hipMemcpy(dev_dataA, dev_dataB, numBytes, hipMemcpyDeviceToDevice);
  }

  timer().endGpuTimer();

  hipMemcpy(odata, dev_dataA, numBytes, hipMemcpyDeviceToHost);
  checkCUDAError("hipMemcpy: dev_dataA -> odata failed!");

  // Convert from inclusive scan to exclusive
  if constexpr (useExclusiveScan) {
    for (int i = n - 1; i > 0; --i) {
      odata[i] = odata[i - 1];
    }
    odata[0] = 0;
  }

  hipFree(dev_dataA);
  checkCUDAError("hipFree: dev_dataA failed!");
  hipFree(dev_dataB);
  checkCUDAError("hipFree: dev_dataB failed!");
}

}  // namespace Naive
}  // namespace StreamCompaction
