#include "hip/hip_runtime.h"
#include "common.h"
#include "naive.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace StreamCompaction {
namespace Naive {

using StreamCompaction::Common::PerformanceTimer;

/// Number of threads per block.
constexpr int blockSize = 256;

/// Whether to return an inclusive or exclusive scan.
constexpr bool useExclusiveScan = true;

PerformanceTimer& timer() {
  static PerformanceTimer timer;
  return timer;
}

__global__ void kernSumPairsForIteration(int n, const int* in, int* out, int stride) {
  int k = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (k >= n) return;

  int outIndex = stride + k;

  out[outIndex] = in[k] + in[outIndex];
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int* odata, const int* idata) {
  int* dev_dataA = nullptr;
  int* dev_dataB = nullptr;

  size_t numBytes = n * sizeof(int);
  hipMalloc(reinterpret_cast<void**>(&dev_dataA), numBytes);
  checkCUDAError("hipMalloc: dev_dataA failed!");
  hipMalloc(reinterpret_cast<void**>(&dev_dataB), numBytes);
  checkCUDAError("hipMalloc: dev_dataB failed!");

  hipMemcpy(dev_dataA, idata, numBytes, hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy: idata -> dev_dataA failed!");
  hipMemcpy(dev_dataB, dev_dataA, numBytes, hipMemcpyDeviceToDevice);
  checkCUDAError("hipMemcpy: dev_dataA -> dev_dataB failed!");

  timer().startGpuTimer();

  for (int iteration = 1; iteration <= ilog2ceil(n); ++iteration) {
    int stride = 1 << (iteration - 1);
    int numDispatches = n - stride;
    int numBlocks = (numDispatches + blockSize - 1) / blockSize;

    kernSumPairsForIteration<<<numBlocks, blockSize>>>(numDispatches, dev_dataA, dev_dataB, stride);

    // Write new results back into A to be read from
    hipMemcpy(dev_dataA, dev_dataB, numBytes, hipMemcpyDeviceToDevice);
  }

  timer().endGpuTimer();

  hipMemcpy(odata, dev_dataA, numBytes, hipMemcpyDeviceToHost);
  checkCUDAError("hipMemcpy: dev_dataA -> odata failed!");

  // Convert from inclusive scan to exclusive
  if constexpr (useExclusiveScan) {
    for (int i = n - 1; i > 0; --i) {
      odata[i] = odata[i - 1];
    }
    odata[0] = 0;
  }

  hipFree(dev_dataA);
  checkCUDAError("hipFree: dev_dataA failed!");
  hipFree(dev_dataB);
  checkCUDAError("hipFree: dev_dataB failed!");
}

}  // namespace Naive
}  // namespace StreamCompaction
